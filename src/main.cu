#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <iostream>

#include "object/hittable.h"
#include "object/sphere.h"
#include "utility/ray.h"
#include "utility/vec3.h"

// limited version of CheckCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define CHECK_CUDA_ERRORS(val) CheckCuda((val), #val, __FILE__, __LINE__)

void CheckCuda(hipError_t result, char const* const func,
               const char* const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
              << file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

__device__ double HitSphere(const Vec3& center, double radius, const Ray& r) {
  Vec3 oc = r.Origin() - center;
  double a = Dot(r.Direction(), r.Direction());
  double b = 2.0f * Dot(oc, r.Direction());
  double c = Dot(oc, oc) - radius * radius;
  double discriminant = b * b - 4 * a * c;
  if (discriminant < 0) {
    return -1.0;
  } else {
    return (-b - std::sqrt(discriminant)) / (2.0 * a);
  }
}

__device__ Color RayColor(const Ray& r) {
  auto t = HitSphere({0, 0, -1}, 0.5, r);
  if (t > 0.0) {
    Vec3 normal = UnitVector(r.At(t) - Vec3(0, 0, -1));
    return 0.5 * Color(normal.X() + 1, normal.Y() + 1, normal.Z() + 1);
  }
  Vec3 unit_direction = UnitVector(r.Direction());
  t = 0.5 * (unit_direction.Y() + 1.0);
  return (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
}

__global__ void Render(Vec3* fb, int max_x, int max_y, Vec3 lower_left_corner,
                       Vec3 horizontal, Vec3 vertical, Vec3 origin) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  double u = double(i) / double(max_x);
  double v = double(j) / double(max_y);
  Ray r(origin, lower_left_corner + u * horizontal + v * vertical);

  unsigned int pixel_index = j * max_x + i;
  fb[pixel_index] = RayColor(r);
}

int main() {
  int image_width = 1200;
  int image_height = 600;
  int tx = 8;
  int ty = 8;

  std::cerr << "Rendering a " << image_width << "x" << image_height
            << " image ";
  std::cerr << "in " << tx << "x" << ty << " blocks.\n";

  int num_pixels = image_width * image_height;
  size_t fb_size = num_pixels * sizeof(Vec3);

  // allocate FB
  Vec3* fb;
  CHECK_CUDA_ERRORS(hipMallocManaged((void**)&fb, fb_size));

  clock_t start, stop;
  start = clock();
  // Render our buffer
  dim3 blocks(image_width / tx + 1, image_height / ty + 1);
  dim3 threads(tx, ty);
  Render<<<blocks, threads>>>(fb, image_width, image_height,
                              Vec3(-2.0, -1.0, -1.0), Vec3(4.0, 0.0, 0.0),
                              Vec3(0.0, 2.0, 0.0), Vec3(0.0, 0.0, 0.0));
  CHECK_CUDA_ERRORS(hipGetLastError());
  CHECK_CUDA_ERRORS(hipDeviceSynchronize());

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  std::string scene_name = "Sphere";
  // Output
  std::ofstream ofs(scene_name + ".ppm");
  // Output FB as Image
  ofs << "P3\n" << image_width << " " << image_height << "\n255\n";
  for (int j = image_height - 1; j >= 0; j--) {
    for (int i = 0; i < image_width; i++) {
      size_t pixel_index = j * image_width + i;
      int ir = int(255.99 * fb[pixel_index].R());
      int ig = int(255.99 * fb[pixel_index].G());
      int ib = int(255.99 * fb[pixel_index].B());
      ofs << ir << " " << ig << " " << ib << "\n";
    }
  }

  CHECK_CUDA_ERRORS(hipFree(fb));
}
